#include "hip/hip_runtime.h"
#include<stdio.h>
#include <bits/stdc++.h>
#include<hip/hip_runtime.h>
#include <thrust/reduce.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>

using namespace std;


void get_adj_matrix(float* graph, int n, float d, FILE *inputFilePtr ){

    if ( inputFilePtr == NULL )  {
        printf( "input.txt file failed to open." );
        return ;
    }

    int m, indexing;
    
    fscanf(inputFilePtr, "%d", &m);
    fscanf(inputFilePtr, "%d", &indexing);

    
    for(int i = 0; i< n ; i++){
    
        for(int j = 0; j< n; ++j){
            graph[i* n + j] = (1 - d)/float(n);
        }
    }

    while(m--){
        int source, destin;
        fscanf(inputFilePtr, "%d", &source);
        fscanf(inputFilePtr, "%d", &destin);
        if (indexing == 0){
            graph[destin* n + source] += d* 1.0  ;
        }
        else{
            graph[(destin - 1)* n + source - 1] += d* 1.0;
        }
    }
}

__global__ void manage_adj_matrix(float* gpu_graph, int n){
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < n){
        float sum = 0.0;

        for (int i = 0; i< n; ++i){
            sum += gpu_graph[i* n + id];
        }

        for (int i = 0; i < n; ++i){
            if (sum != 0.0){
                gpu_graph[i* n + id] /= sum;
            }
            else{
                gpu_graph[i* n + id] = (1/(float)n);
            }
        }
    }
}

__global__ void initialize_rank(float* gpu_r, int n){
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if(id < n){
        gpu_r[id] = (1/(float)n);
    }
}

__global__ void store_rank(float* gpu_r,float* gpu_r_last, int n){
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if(id < n){
        gpu_r_last[id] = gpu_r[id];
    }
}

__global__ void matmul(float* gpu_graph, float* gpu_r, float* gpu_r_last, int n){
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if(id < n){
        float sum = 0.0;

        for (int j = 0; j< n; ++j){
            sum += gpu_r_last[j] * gpu_graph[id* n + j];
        }

        gpu_r[id] = sum;
    }
}

__global__ void rank_diff(float* gpu_r,float* gpu_r_last, int n){
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if(id < n){
        gpu_r_last[id] = abs(gpu_r_last[id] - gpu_r[id]);
    }
}

__global__ void init_pair_array(pair<float, int>* gpu_r_nodes, float * gpu_r, int n){
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if(id < n){
        gpu_r_nodes[id].first = gpu_r[id];
        gpu_r_nodes[id].second = id + 1;
    }
}


void power_method(float *graph, float *r, int n, int nblocks, int BLOCKSIZE, int max_iter = 1000, float eps = 0.000001 ){
   
    float* r_last = (float*) malloc(n * sizeof(float));
    
    float* gpu_graph;
    hipMalloc(&gpu_graph, sizeof(float)*n*n);
    hipMemcpy(gpu_graph, graph, sizeof(float)*n*n, hipMemcpyHostToDevice);

    float* gpu_r;
    hipMalloc(&gpu_r, sizeof(float)*n);
    //hipMemcpy(gpu_r, r, sizeof(float)*n, hipMemcpyHostToDevice);

    float* gpu_r_last;
    hipMalloc(&gpu_r_last, sizeof(float)*n);
    //hipMemcpy(gpu_r_last, r_last, sizeof(float)*n, hipMemcpyHostToDevice);



    initialize_rank<<<nblocks, BLOCKSIZE>>>(gpu_r, n);
    hipDeviceSynchronize();



    while(max_iter--){

        store_rank<<<nblocks, BLOCKSIZE>>>(gpu_r, gpu_r_last, n);
        hipDeviceSynchronize();

        matmul<<<nblocks, BLOCKSIZE>>>(gpu_graph, gpu_r, gpu_r_last, n);
        hipDeviceSynchronize();
        
        rank_diff<<<nblocks, BLOCKSIZE>>>(gpu_r, gpu_r_last, n);
        hipDeviceSynchronize();

        hipMemcpy(r_last, gpu_r_last, n* sizeof(float), hipMemcpyDeviceToHost);
        float result = thrust::reduce( r_last, r_last + n);

        if(result < eps){
            hipMemcpy(r, gpu_r, n* sizeof(float), hipMemcpyDeviceToHost);
            return;
        }
    }
    hipMemcpy(r, gpu_r, n* sizeof(float), hipMemcpyDeviceToHost);
    return;
}

void top_nodes(float* r, int n, int nblocks, int BLOCKSIZE, int count = 10){

    pair<float, int> *r_nodes = (pair<float, int> *) malloc ( n * sizeof (pair<float, int>) );
    pair<float, int> *gpu_r_nodes;

    hipMalloc(&gpu_r_nodes, n * sizeof (pair<float, int>));

    float* gpu_r;
    hipMalloc(&gpu_r, sizeof(float)*n);
    hipMemcpy(gpu_r, r, sizeof(float)*n, hipMemcpyHostToDevice);

    init_pair_array<<<nblocks, BLOCKSIZE>>>(gpu_r_nodes, gpu_r, n);

    hipMemcpy(r_nodes, gpu_r_nodes, n * sizeof (pair<float, int>), hipMemcpyDeviceToHost);

    thrust::sort(thrust::host, r_nodes, r_nodes + n);

    int rank =1;
    while(rank <= count){
        printf("Rank %d Node is %d\n", rank, r_nodes[n - rank].second);
        rank++;
    }
}

int main(int argc, char** argv){

    clock_t start, end;

    FILE *inputFilePtr;

    char * inputfile = argv[1];

    char * bsize = argv[2];
    int BLOCKSIZE = atoi(bsize);

    inputFilePtr = fopen(inputfile, "r");

    int n; 

    fscanf(inputFilePtr, "%d", &n);
    int nblocks = ceil(float(n) / BLOCKSIZE);

    float* graph = (float*)malloc(n*n*sizeof(float));
    float* r = (float*) malloc(n * sizeof(float));

    float d = 0.85;

    get_adj_matrix(graph, n, d, inputFilePtr);

    float* gpu_graph;
    hipMalloc(&gpu_graph, sizeof(float)*n*n);
    hipMemcpy(gpu_graph, graph, sizeof(float)*n*n, hipMemcpyHostToDevice);
    
    start = clock();

    manage_adj_matrix<<<nblocks, BLOCKSIZE>>>(gpu_graph, n);
    hipMemcpy(graph, gpu_graph, sizeof(float)*n*n, hipMemcpyDeviceToHost);

    power_method(graph, r, n, nblocks, BLOCKSIZE );

    top_nodes(r, n, nblocks, BLOCKSIZE);

    end = clock();

    printf("Time taken :%f for parallel implementation with %d nodes.\n", float(end - start), n);
    return 0;
}